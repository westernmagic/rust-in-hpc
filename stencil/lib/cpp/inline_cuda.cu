#include "hip/hip_runtime.h"
#include "array.hpp"
#include <cassert>
#include <cmath>
#include <cstddef>

// using cooperative group API leads to weird errors

__global__
void diffuse_inline_cuda_kernel(
	float *     const in_field,
	float *     const out_field,
	std::size_t const nx,
	std::size_t const ny,
	std::size_t const nz,
	std::size_t const num_halo,
	float       const alpha
) {
	assert(in_field != nullptr);
	assert(out_field != nullptr);
	assert(nx > 0);
	assert(ny > 0);
	assert(nz > 0);
	assert(num_halo > 0);
	assert(!std::isnan(alpha));

	ArrayView3 in_field_(  in_field,   nx + 2 * num_halo, ny + 2 * num_halo, nz);
	ArrayView3 out_field_( out_field,  nx + 2 * num_halo, ny + 2 * num_halo, nz);

	float const alpha_20 = -20 * alpha + 1;
	float const alpha_08 =   8 * alpha;
	float const alpha_02 =  -2 * alpha;
	float const alpha_01 =  -1 * alpha;

	std::size_t const i = blockDim.x * blockIdx.x + threadIdx.x;
	std::size_t const j = blockDim.y * blockIdx.y + threadIdx.y;
	std::size_t const k = blockDim.z * blockIdx.z + threadIdx.z;

	if (
		0        <= k && k < nz            &&
		num_halo <= j && j < ny + num_halo &&
		num_halo <= i && i < nx + num_halo
	) {
		out_field_(i, j, k) =
			  alpha_20 * in_field_(i,     j,     k)  
			+ alpha_08 * in_field_(i - 1, j,     k)  
			+ alpha_08 * in_field_(i + 1, j,     k)  
			+ alpha_08 * in_field_(i,     j - 1, k)
			+ alpha_08 * in_field_(i,     j + 1, k)
			+ alpha_02 * in_field_(i - 1, j - 1, k)
			+ alpha_02 * in_field_(i - 1, j + 1, k)
			+ alpha_02 * in_field_(i + 1, j - 1, k)
			+ alpha_02 * in_field_(i + 1, j + 1, k)
			+ alpha_01 * in_field_(i - 2, j,     k)  
			+ alpha_01 * in_field_(i + 2, j,     k)  
			+ alpha_01 * in_field_(i,     j - 2, k)
			+ alpha_01 * in_field_(i,     j + 2, k)
		;  
	}
}

extern "C" __host__ void diffuse_inline_cuda(
	float *     const in_field,
	float *     const out_field,
	std::size_t const nx,
	std::size_t const ny,
	std::size_t const nz,
	std::size_t const num_halo,
	float       const alpha,
	std::size_t const num_iter
) noexcept {
	using std::swap;

	assert(in_field != nullptr);
	assert(out_field != nullptr);
	assert(nx > 0);
	assert(ny > 0);
	assert(nz > 0);
	assert(num_halo > 0);
	assert(!std::isnan(alpha));
	assert(num_iter > 0);

	std::size_t const size = (nx + 2 * num_halo) * (ny + 2 * num_halo) * nz * sizeof(float);
	float * in_field_d   = nullptr;
	float * out_field_d  = nullptr;
	float * tmp1_field_d = nullptr;

	hipMalloc((void**)&in_field_d,   size);
	assert(hipGetLastError() == hipSuccess);
	hipMalloc((void**)&out_field_d,  size);
	assert(hipGetLastError() == hipSuccess);
	hipMalloc((void**)&tmp1_field_d, size);
	assert(hipGetLastError() == hipSuccess);

	hipMemcpy(in_field_d, in_field, size, hipMemcpyHostToDevice);
	assert(hipGetLastError() == hipSuccess);

	dim3 const blockDim(1, 1, 1);
	dim3 const gridDim((nx + 2 * num_halo + blockDim.x - 1) / blockDim.x, (ny + 2 * num_halo + blockDim.y - 1) / blockDim.y, (nz + blockDim.z - 1) / blockDim.z);

	assert(0 < gridDim.x  && gridDim.x  <= (1 << 31) - 1);
	assert(0 < gridDim.y  && gridDim.y  <= 65535);
	assert(0 < gridDim.z  && gridDim.z  <= 65535);
	assert(0 < blockDim.x && blockDim.x <= 1024);
	assert(0 < blockDim.y && blockDim.y <= 1024);
	assert(0 < blockDim.z && blockDim.z <= 64);

	for (std::size_t iter = 0; iter < num_iter; ++iter) {
		// Intel/Cray generate invalid stubs, pass all 0 for gridDim, blockDim and bad pointers for args
		diffuse_inline_cuda_kernel<<<gridDim, blockDim>>>(in_field_d, out_field_d, nx, ny, nz, num_halo, alpha);
		assert(hipPeekAtLastError() == hipSuccess);
		hipDeviceSynchronize();
		assert(hipGetLastError() == hipSuccess);
		swap(in_field_d, out_field_d);
	}

	hipMemcpy(out_field, out_field_d, size, hipMemcpyDeviceToHost);
	assert(hipGetLastError() == hipSuccess);

	hipFree(tmp1_field_d);
	assert(hipGetLastError() == hipSuccess);
	tmp1_field_d = nullptr;
	hipFree(out_field_d);
	assert(hipGetLastError() == hipSuccess);
	out_field_d = nullptr;
	hipFree(in_field_d);
	assert(hipGetLastError() == hipSuccess);
	in_field_d = nullptr;
}
